#include "hip/hip_runtime.h"
#define WINDOWS 1

#ifdef WINDOWS
// Import these libraries if using MS Visual Studio for development.
// They are needed by nvcc to interface with MS Visual Studio.
#include "hip/hip_runtime.h"
#include ""
#endif // !WINDOWS

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define SIZE 1024       // DEFAULT values for a, b and c
#define DATATYPE int    // datatype used. For getting GFLOPS use float.
#define MAX_RANDOM 20   // Random limit
#define BLOCK_SIZE 32   // DEFAULT block size

/**
 * Kernel for multiplication of matrices
 * Each thread calculates the element of the resultant matrix
 * The thread decides the element using block_index and thread_index values
 * Since it calculates the resultant with help of grid, I did not need a call to __syncThreads()
 */
__global__ 
void multiply(DATATYPE *p, DATATYPE *q, DATATYPE *r, int a, int b, int c) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;    // get the row index for resultant matrix
    int col = blockDim.x * blockIdx.x + threadIdx.x;    // get the column index for column matrix
    int sum = 0, i = 0;
    if (row < a && col < c) {
        for (i = 0; i < b; ++i) {
            sum += p[row * b + i] * q[i * c + col];
        }
        r[row * c + col] = sum;
    }
    
}

int main(int argc, char *argv[]) {
    // declarations
    int a = SIZE, b = SIZE, c = SIZE;
    DATATYPE *p, *q, *r, *hr;
    int i, j, k;
    float time_elapsed = 0;

    DATATYPE *d_p, *d_q, *d_r;
    int N_BLOCKS = BLOCK_SIZE, N_THREADS;

    hipEvent_t start, stop;

    // read from command line if possible
    if (argc == 4) {
        a = atoi(argv[1]);
        b = atoi(argv[2]);
        c = atoi(argv[3]);
        printf("Reading values of a, b and c\na = %d, b = %d, c = %d\n", a, b, c);
    } else { // otherwise proceed with default values
        fprintf(stderr, "proceeding with default values of a, b and c (1024)\n");
        fprintf(stderr, "usage: kernel a b c\n");
    }
    
    // get number of blocks and threads to use from user
    printf("Enter N_BLOCKS: ");
    scanf("%d", &N_BLOCKS);
    printf("Enter N_THREADS: ");
    scanf("%d", &N_THREADS);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate memory using unified memory model
    hipHostMalloc((void**) &p, sizeof(DATATYPE) * a * b);
    hipHostMalloc((void**) &q, sizeof(DATATYPE) * b * c);
    hipHostMalloc((void**) &r, sizeof(DATATYPE) * a * c);

    // create random variable using seed and initialize matrices
    // Initialize matrix c to 0
    srand(time(NULL));
    for (i = 0; i < a; ++i) {
        for (j = 0; j < b; ++j) {
            p[i * b + j] = rand() % MAX_RANDOM;
        }
    }
    for (i = 0; i < b; ++i) {
        for (j = 0; j < c; ++j) {
            q[i * c + j] = rand() % MAX_RANDOM;
        }
    }
    for (i = 0; i < a; ++i) {
        for (j = 0; j < b; ++j) {
            r[i * c + j] = 0;
        }
    }

    // find the optimal configuration of number of threads
    // create a dim3 structure for it
    unsigned int rows = (a + N_BLOCKS - 1) / N_BLOCKS;
    unsigned int cols = (c + N_BLOCKS - 1) / N_BLOCKS;
    dim3 dimGrid(cols, rows);
    dim3 dimBlock(N_BLOCKS, N_BLOCKS);

    // start the clock
    hipEventRecord(start);
    // allocate device memory
    hipMalloc((void **) &d_p, sizeof(DATATYPE) * a * b);
    hipMalloc((void **) &d_q, sizeof(DATATYPE) * b * c);
    hipMalloc((void **) &d_r, sizeof(DATATYPE) * a * c);

    // copy the contents from host to device
    hipMemcpy(d_p, p, sizeof(DATATYPE) * a * b, hipMemcpyHostToDevice);
    hipMemcpy(d_q, q, sizeof(DATATYPE) * b * c, hipMemcpyHostToDevice);
    hipMemcpy(d_r, r, sizeof(DATATYPE) * a * c, hipMemcpyHostToDevice);

    // call kernel
    multiply <<< dimGrid, dimBlock >>> (d_p, d_q, d_r, a, b, c);
    
    // copy the results back and let all threads sync
    hipMemcpy(r, d_r, sizeof(DATATYPE) * a * c, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    //stop clock and free memory
    hipFree(d_p);
    hipFree(d_q);
    hipFree(d_r);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // find the time taken
    hipEventElapsedTime(&time_elapsed, start, stop);
    // and print it
    printf("Time: %f msecs\n", time_elapsed);

    // free host memory
    hipHostFree(p);
    hipHostFree(q);
    hipHostFree(r);

    return 0;
}
